#include <stdlib.h>
#include <time.h>
#include "update.cuh"
#include "vector2d.cuh"
#include "display_cu.h"
#include "body.h"

//Screen dimensions
const int SCREEN_WIDTH = 1000;
const int SCREEN_HEIGHT = 1000;

int main(int argc, char const *argv[])
{
    int nb_body = 900;
    dim3 grid_th(32, 32);
    //dim3 grid_blk(__max(1, nb_body/16),__max(1, nb_body/16));
    dim3 grid_blk(1 + nb_body/32, 1 + nb_body/32);
    /* Setup for timing */
    struct timespec phX_time_start;
    struct timespec phX_time_end;
    unsigned long long avr_phX_time = 0;
    unsigned int phX_frame_nb = 0;

    /* Setup for PhX */
    body* body_list = (body*) malloc(nb_body*sizeof(body));
    for(int i = 0; i < nb_body; i++){
        body_list[i].pos.x = 33 * (i % 30);
        body_list[i].pos.y = 33 * (i / 30);
        body_list[i].vel.x = 0;
        body_list[i].vel.y = 0;
        body_list[i].mass = 1;
    }

    body* body_list_gpu;
    hipMalloc(&body_list_gpu, nb_body*sizeof(body));
    hipMemcpy(body_list_gpu, body_list, nb_body*sizeof(body), hipMemcpyHostToDevice);
    
    vector2d* k1;
    vector2d* k2;
    vector2d* k3;
    vector2d* k4;
    hipMalloc(&k1, nb_body * sizeof(vector2d));
    hipMalloc(&k2, nb_body * sizeof(vector2d));
    hipMalloc(&k3, nb_body * sizeof(vector2d));
    hipMalloc(&k4, nb_body * sizeof(vector2d));

    /* Setup graphics */
    int paused = 1;
    int running = 1;
    SDL_Renderer* renderer = NULL;
    SDL_Window* window = NULL;
    enum keyboard_action cur_action;
    SDL_Event event;
    SDL_FRect* rects =(SDL_FRect*) malloc(nb_body*sizeof(SDL_FRect));
    start_graphics(SCREEN_WIDTH, SCREEN_HEIGHT, window, &renderer);
    init_frames(body_list, nb_body, rects);
    draw_frame(body_list, nb_body, renderer, rects);

    /* Graphics main loop */
    while(running){
        cur_action = check_keyboard();
        switch (cur_action){
        case EXIT_SIM:
            running = 0;
            end_graphics(window);
            break;
        case TOGGLE_SIM:
            paused = !paused;
            break;
        case NEXT_FRAME:
            if (paused){
                timespec_get(&phX_time_start, TIME_UTC);
                for(int i = 0; i < 2; i++){
                    /* START OF PHYSIC ITERATION */
                    cu_reset_k<<<1 + nb_body/256, 256>>>(k1, k2, k3, k4, nb_body);
                    hipDeviceSynchronize();
                    cu_calc_k1k2<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2);
                    hipDeviceSynchronize();
                    cu_calc_k3k4<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
                    hipDeviceSynchronize();
                    cu_update<<<1 + nb_body/256, 256>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
                    hipDeviceSynchronize();
                    hipMemcpy(body_list, body_list_gpu, nb_body*sizeof(body), hipMemcpyDeviceToHost);
                    /* END OF PHYSIC ITERATION */                
                }
                timespec_get(&phX_time_end, TIME_UTC);
                draw_frame(body_list, nb_body, renderer, rects);
                phX_frame_nb+= 1;
                avr_phX_time += (phX_time_end.tv_sec - phX_time_start.tv_sec)*1000000000 + phX_time_end.tv_nsec-phX_time_start.tv_nsec;
            }
            break;
        default:
            break;
        }
        if(!paused){
            timespec_get(&phX_time_start, TIME_UTC);
            for(int i = 0; i < 2; i++){
                /* START OF PHYSIC ITERATION */
                cu_reset_k<<<1 + nb_body/256, 256>>>(k1, k2, k3, k4, nb_body);
                hipDeviceSynchronize();
                cu_calc_k1k2<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2);
                hipDeviceSynchronize();
                cu_calc_k3k4<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
                hipDeviceSynchronize();
                cu_update<<<1 + nb_body/256, 256>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
                hipDeviceSynchronize();
                hipMemcpy(body_list, body_list_gpu, nb_body*sizeof(body), hipMemcpyDeviceToHost);
                /* END OF PHYSIC ITERATION */                     
            }
            timespec_get(&phX_time_end, TIME_UTC);
            phX_frame_nb += 1;
            avr_phX_time += (phX_time_end.tv_sec - phX_time_start.tv_sec)*1000000000 + phX_time_end.tv_nsec-phX_time_start.tv_nsec;
            draw_frame(body_list, nb_body, renderer, rects);
        }
    }

    // /* START OF PHYSIC ITERATION */
    // cu_reset_k<<<__max(1, nb_body/256), 256>>>(k1, k2, k3, k4, nb_body);
    // hipDeviceSynchronize();
    // cu_calc_k1k2<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2);
    // hipDeviceSynchronize();
    // cu_calc_k3k4<<<grid_blk, grid_th>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
    // hipDeviceSynchronize();
    // cu_update<<<__max(1, nb_body/256), 256>>>(body_list_gpu, nb_body, 1., k1, k2, k3, k4);
    // hipDeviceSynchronize();
    // hipMemcpy(body_list, body_list_gpu, nb_body*sizeof(body), hipMemcpyDeviceToHost);
    // /* END OF PHYSIC ITERATION */
    printf("Average frame duration : %lld microseconds over %d frames \n", (avr_phX_time/phX_frame_nb)/1000 , phX_frame_nb);

    hipFree(k1);
    hipFree(k2);
    hipFree(k3);
    hipFree(k4);
    hipFree(body_list_gpu);
    free(body_list);

    return 0;
}
