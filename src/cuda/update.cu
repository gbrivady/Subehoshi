#include "hip/hip_runtime.h"
#include "update.cuh"

__device__
void cu_calc_acc(body* body1, body* body2, vector2d* output_v2d, vector2d* pos_offset){
    v2d_add(&(body1->pos), pos_offset, output_v2d);
    v2d_sub(&(body2->pos), output_v2d, output_v2d);
    double strength = body1->mass * body2->mass *
    pow(fmax(v2d_norm2(output_v2d), CU_MINIMAL_DISTANCE_SQ), -3.0/2.0);    
    v2d_scale(strength, output_v2d);
}

__global__
void cu_calc_k1k2(body* body_list, const int nb_body, const double timestep,
vector2d* k1, vector2d* k2){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= nb_body || j >= nb_body || i == j)
        return;

    vector2d pos_offset = {0., 0.};
    vector2d temp_k = {0., 0.};
    cu_calc_acc(&body_list[i], &body_list[j], &temp_k, &pos_offset);

    atomicAdd(&(k1[i].x), temp_k.x);
    atomicAdd(&(k1[i].y), temp_k.y);

    pos_offset.x = (body_list[i].vel.x) * timestep / 2. ;
    pos_offset.y = (body_list[i].vel.y) * timestep / 2. ;

    cu_calc_acc(&body_list[i], &body_list[j], &temp_k, &pos_offset);

    atomicAdd(&(k2[i].x), temp_k.x);
    atomicAdd(&(k2[i].y), temp_k.y);
}

__global__ 
void cu_calc_k3k4(body* body_list, const int nb_body, const double timestep,
vector2d* k1, vector2d* k2, vector2d* k3, vector2d* k4){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= nb_body || j >= nb_body)
        return;

    vector2d pos_offset = {0., 0.};
    vector2d temp_k = {0., 0.};

    pos_offset.x = (body_list[i].vel.x) * timestep / 2. ;
    pos_offset.y = (body_list[i].vel.y) * timestep / 2. ;

    pos_offset.x += (k1[i].x) * (timestep*timestep) / 4. ;
    pos_offset.y += (k1[i].y) * (timestep*timestep) / 4. ;

    cu_calc_acc(&body_list[i], &body_list[j], &temp_k, &pos_offset);

    atomicAdd(&(k3[i].x), temp_k.x);
    atomicAdd(&(k3[i].y), temp_k.y);

    pos_offset.x = (body_list[i].vel.x) * timestep;
    pos_offset.y = (body_list[i].vel.y) * timestep;

    pos_offset.x += (k2[i].x) * (timestep*timestep) / 2. ;
    pos_offset.y += (k2[i].y) * (timestep*timestep) / 2. ;

    cu_calc_acc(&body_list[i], &body_list[j], &temp_k, &pos_offset);

    atomicAdd(&(k4[i].x), temp_k.x);
    atomicAdd(&(k4[i].y), temp_k.y);
}

__global__ 
void cu_update(body* body_list, const int nb_body, const double timestep,
vector2d* k1, vector2d* k2, vector2d* k3, vector2d* k4){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= nb_body)
        return;

    vector2d temp_v2d;

    v2d_copy(&temp_v2d, &(body_list[i].vel));
    v2d_scale(timestep, &temp_v2d);
    v2d_incr(&(body_list[i].pos), &temp_v2d);

    v2d_copy(&temp_v2d, &k1[i]);
    v2d_incr(&temp_v2d, &k2[i]);
    v2d_incr(&temp_v2d, &k3[i]);
    v2d_scale(timestep*timestep/6.0, &temp_v2d);
    v2d_incr(&(body_list[i].pos), &temp_v2d);

    v2d_copy(&temp_v2d, &k2[i]);
    v2d_incr(&temp_v2d, &k3[i]);
    v2d_scale(2, &temp_v2d);

    v2d_incr(&temp_v2d, &k1[i]);
    v2d_incr(&temp_v2d, &k4[i]);
    v2d_scale(timestep/6.0, &temp_v2d);
    v2d_incr(&(body_list[i].vel), &temp_v2d);
    
}

__global__
void cu_reset_k(vector2d* k1, vector2d* k2, vector2d* k3, vector2d* k4, const int nb_body){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= nb_body)
        return;
    k1[i].x = 0;
    k1[i].y = 0;
    k2[i].x = 0;
    k2[i].y = 0;
    k3[i].x = 0;
    k3[i].y = 0;
    k4[i].x = 0;
    k4[i].y = 0;
}
