#include "hip/hip_runtime.h"
#include "vector2d.cuh"

__device__
void v2d_reset(vector2d* u){
    u->x = 0;
    u->y = 0;
}

__device__
void v2d_copy(vector2d* v2dp, vector2d* v2dc){
    v2dp->x = v2dc->x;
    v2dp->y = v2dc->y;
}

__device__
void v2d_incr(vector2d* u, vector2d* v){
    u->x = u->x + v->x;
    u->y = u->y + v->y;
}

__device__
void v2d_add(vector2d * u, vector2d * v, vector2d* w){
    w->x = u->x + v->x;
    w->y = u->y + v->y;
}

__device__
void v2d_decr(vector2d* u, vector2d* v){
    u->x = u->x - v->x;
    u->y = u->y - v->y;
}

__device__
void v2d_sub(vector2d * u, vector2d * v, vector2d* w){
    w->x = u->x - v->x;
    w->y = u->y - v->y;
}

__device__
void v2d_scale(double k, vector2d* u){
    u->x = k*u->x;
    u->y = k*u->y;
}

__device__
void v2d_normalise(vector2d* u){
    double norm = v2d_norm(u);
    u->x = u->x/norm;
    u->y = u->y/norm;
}

__device__
double v2d_norm2(vector2d* u){
    return u->x*u->x + u->y*u->y;
}
__device__
double v2d_norm(vector2d* u){
    double norm = u->x*u->x + u->y*u->y;
    return sqrt(norm);
}
